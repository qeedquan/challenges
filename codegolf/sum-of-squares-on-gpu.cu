/*

Write a program or function that:

Takes as input a list or array of 128 numbers.

Sends that array to a GPU

Performs a calculation on the GPU to square each element of the array and then sum them

Gets the answer from the GPU, and returns or displays it

So your program will simply return the sum of the squares of its input array, but it will perform the calculation on a GPU.

The input numbers should be sent to the GPU in the form of signed or unsigned integer types or floats, of 32 or more bits in size.

You may make any assumptions you like about hardware, e.g. you may assume a CUDA or OpenCl capable card is present without checking for it.

This is code-golf. The smallest code (in bytes) will be considered the winner for each language.

*/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 128

__global__ void
sumofsquares(float *output, float *input, int length)
{
	float value;
	int i;

	value = 0;
	for (i = 0; i < length; i++)
		value += input[i] * input[i];
	*output = value;
}

int
main()
{
	float *input;
	float *output;
	float values[SIZE];
	float result;
	int status;
	int i;

	status = 0;
	input = output = NULL;
	if (hipMalloc(&input, sizeof(*input) * SIZE) != hipSuccess)
		goto error;

	if (hipMalloc(&output, sizeof(*output)) != hipSuccess)
		goto error;

	for (i = 0; i < SIZE; i++)
		values[i] = i;

	hipMemcpy(input, values, sizeof(values), hipMemcpyHostToDevice);

	sumofsquares<<<1, 1>>>(output, input, SIZE);
	hipMemcpy(&result, output, sizeof(result), hipMemcpyDeviceToHost);

	printf("%f\n", result);

	if (0) {
	error:
		status = 1;
	}

	hipFree(input);
	hipFree(output);
	return status;
}
